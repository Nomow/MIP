#include "hip/hip_runtime.h"
#include "morphology2d.cuh"
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <vector>
#include <nppdefs.h>
#include <npp.h>
#include <nppi.h>

/**
  cuda error checking helper methods for data copy between host and device and device and host and kernel error check.
*/
#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif
    return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif
    return;
}

/**
  allocates memory in device and copies data from host memory to padded device memory.
*/
__host__ void ConvertHostToDevice(uint8_t *inHostSrc, uint32_t inSrcWidth, uint32_t inSrcHeight, uint8_t *&outDeviceDst, size_t &outDstStep) {
  CudaSafeCall(hipMallocPitch(&outDeviceDst, &outDstStep, inSrcWidth * sizeof(uint8_t), inSrcHeight));
  CudaSafeCall(hipMemcpy2D(outDeviceDst, outDstStep, inHostSrc,  inSrcWidth * sizeof(uint8_t),
                            inSrcWidth * sizeof(uint8_t), inSrcHeight, hipMemcpyHostToDevice));
}

/**
  allocates memory in device and copies data from host memory to padded device memory.
*/
__host__ void ConvertHostToDevice(uint8_t *inHostSrc, uint32_t inSrcSize, uint8_t *&outDeviceDst) {
  CudaSafeCall(hipMalloc(&outDeviceDst, inSrcSize * sizeof(uint8_t)));
  CudaSafeCall(hipMemcpy(outDeviceDst, inHostSrc,  inSrcSize * sizeof(uint8_t), hipMemcpyHostToDevice));
}

/**
  allocates memory in host and copies data from device memory to host memory.
*/
__host__ void ConvertDeviceToHost(uint8_t *inDeviceSrc, size_t inSrcStep, uint32_t inSrcWidth, uint32_t inSrcHeight, uint8_t *&outHostDst) {
    outHostDst = (uint8_t*)malloc(inSrcWidth * inSrcHeight);
    CudaSafeCall(hipMemcpy2D(outHostDst, inSrcWidth * sizeof(uint8_t), inDeviceSrc,
                              inSrcStep, inSrcWidth * sizeof(uint8_t), inSrcHeight, hipMemcpyDeviceToHost));
}

/**
  calculates center of kernel
*/
__host__ void KernelCenter(NppiSize inKernelSize, NppiPoint &outCenter) {
  outCenter.x = inKernelSize.width / 2;
  outCenter.y = inKernelSize.height / 2;
}

/**
  calculates radius of kernel
*/
__host__ void KernelRadius(NppiSize inKernelSize, NppiSize &outRadius) {
  outRadius.width = (inKernelSize.width - 1) / 2;
  outRadius.height = (inKernelSize.height - 1) / 2;
}

/**
  adds border to border image
*/
__host__ void AddConstBorderToImage(uint8_t *inDeviceSrc, size_t inSrcstep, NppiSize inSrcSize, NppiSize inPadRadius,
                                    uint8_t inPadValue, uint8_t *&outDeviceDst, size_t &outDstStep, NppiSize &outDstSize) {

  // calculates new size
  uint32_t width = inSrcSize.width + 2 * inPadRadius.width;
  uint32_t height = inSrcSize.height + 2 * inPadRadius.height;
  outDstSize.width = width;
  outDstSize.height = height;

  // allocates memory for output
  CudaSafeCall(hipMallocPitch(&outDeviceDst, &outDstStep, width * sizeof(uint8_t), height));

  // adds paddibng to Src
  nppiCopyConstBorder_8u_C1R(inDeviceSrc, inSrcstep, inSrcSize, outDeviceDst, outDstStep,
                             outDstSize, inPadRadius.height, inPadRadius.width, inPadValue);
}

/**
  erosion of 2d image with cons
*/
void Erode(uint8_t *inSrc, uint32_t inSrcWidth, uint32_t inSrcHeight, uint8_t* inKernel, uint8_t inKernelWidth, uint8_t inKernelHeight, uint8_t *&outDst) {
  const uint8_t foregroundValue = 1;
  // copies src to device memory
  size_t deviceSrcStep;
  uint8_t* deviceSrc;
  ConvertHostToDevice(inSrc, inSrcWidth, inSrcHeight, deviceSrc, deviceSrcStep);
  NppiSize srcSize = {inSrcWidth, inSrcHeight};

  // copies kernel to device memory
  uint8_t* deviceKernel;
  uint32_t kernelTotalPixels = inKernelWidth * inKernelHeight;
  ConvertHostToDevice(inKernel, kernelTotalPixels, deviceKernel);
  NppiSize kernelSize = {inKernelWidth, inKernelHeight};

  // adds padding to src for border control
  NppiSize kernelRadius;
  KernelRadius(kernelSize, kernelRadius);
  size_t paddedDeviceSrcStep;
  uint8_t* paddedDeviceSrc;
  NppiSize paddedDeviceSrcSize;
  AddConstBorderToImage(deviceSrc, deviceSrcStep, srcSize, kernelRadius, foregroundValue, paddedDeviceSrc, paddedDeviceSrcStep, paddedDeviceSrcSize);
  paddedDeviceSrc = paddedDeviceSrc + kernelRadius.height * paddedDeviceSrcStep + kernelRadius.width * sizeof(uint8_t);
  // allocates memory for output
  size_t deviceDstStep;
  uint8_t *deviceDst;
  CudaSafeCall(hipMallocPitch(&deviceDst, &deviceDstStep, inSrcWidth * sizeof(uint8_t), inSrcHeight));

  NppiPoint kernelCenter;
  KernelCenter(kernelSize, kernelCenter);

  nppiErode_8u_C1R(paddedDeviceSrc, paddedDeviceSrcStep, deviceDst, deviceDstStep, srcSize, deviceKernel, kernelSize, kernelCenter);

  CudaCheckError();
  ConvertDeviceToHost(deviceDst, deviceDstStep, inSrcWidth, inSrcHeight, outDst);
  hipFree(deviceDst);
  hipFree(deviceKernel);
}


/**
  2d erosion of built in npp erode with border control
*/
void ErodeWithBorderControl(uint8_t *inSrc, uint32_t inSrcWidth, uint32_t inSrcHeight, uint8_t* inKernel, uint8_t inKernelWidth, uint8_t inKernelHeight, uint8_t *&outDst) {

  // copies src to device memory
  size_t deviceSrcStep;
  uint8_t* deviceSrc;
  ConvertHostToDevice(inSrc, inSrcWidth, inSrcHeight, deviceSrc, deviceSrcStep);
  NppiSize srcSize = {inSrcWidth, inSrcHeight};

  // copies kernel to device memory
  uint8_t* deviceKernel;
  uint32_t kernelPixelCount = inKernelWidth * inKernelHeight;
  ConvertHostToDevice(inKernel, kernelPixelCount, deviceKernel);
  NppiSize kernelSize = {inKernelWidth, inKernelHeight};

  // allocates memory for output
  size_t deviceDstStep;
  uint8_t *deviceDst;
  CudaSafeCall(hipMallocPitch(&deviceDst, &deviceDstStep, inSrcWidth * sizeof(uint8_t), inSrcHeight));

  NppiPoint kernelCenter;
  KernelCenter(kernelSize, kernelCenter);

 	NppiPoint offset = {0, 0};

  // erosion
  nppiErodeBorder_8u_C1R(deviceSrc, deviceSrcStep, srcSize, offset, deviceDst,
                         deviceDstStep, srcSize, deviceKernel, kernelSize, kernelCenter, NPP_BORDER_REPLICATE);
  CudaCheckError();
  ConvertDeviceToHost(deviceDst, deviceDstStep, inSrcWidth, inSrcHeight, outDst);
  hipFree(deviceDst);
  hipFree(deviceKernel);
}







//
// __device__ int GetGlobal3dThreadIdx() {
//   int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
//   int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
//   return threadId;
// }
//
//
// __device__ bool Erode(int inOrigx, int inOrigy, int inOrigz,
//                       hipPitchedPtr inImg, int inPaddingx, int inPaddingy, int inPaddingz,
//                       hipPitchedPtr inGpuStructElem, int inStructElemRadiusx, int inStructElemRadiusy, int inStructElemRadiusz,
//                       int *inGpuStructElemForegroundDataPosx, int *inGpuStructElemForegroundDataPosy, int *inGpuStructElemForegroundDataPosz, int kernelDimSize) {
//
//     int x = inOrigx + inPaddingx - inStructElemRadiusx;
//     int y = inOrigy + inPaddingy - inStructElemRadiusy;
//     int z = inOrigz + inPaddingz - inStructElemRadiusz;
//
//     for (int i = 0; i < kernelDimSize; ++i) {
//
//       // gets kernel pixel
//       int kx = *(inGpuStructElemForegroundDataPosx + i);
//       int ky = *(inGpuStructElemForegroundDataPosy + i);
//       int kz = *(inGpuStructElemForegroundDataPosz + i);
//       unsigned char kernelPixel = getPixel(inGpuStructElem, kx, ky, kz);
//
//       // gets image pixel
//       int imx = x + kx;
//       int imy = y + ky;
//       int imz = z + kz;
//       unsigned char imagePixel = getPixel(inImg, imx, imy, imz);
//
//       if (!(kernelPixel && imagePixel)) {
//         return true;
//       }
//     }
//     return false;and
// }
//
//
// __global__ void ImgErosion(hipPitchedPtr inGpuOrigImg, int *inGpuOrigImgForegoundDataPosx, int *inGpuOrigImgForegoundDataPosy,  int *inGpuOrigImgForegoundDataPosz, int inImgDimSize,
//                            hipPitchedPtr inPaddedImg, int inPaddingx, int inPaddingy, int inPaddingz,
//                            hipPitchedPtr inGpuStructElem, int inStructElemRadiusx, int inStructElemRadiusy, int inStructElemRadiusz,
//                            int *inGpuStructElemForegroundDataPosx, int *inGpuStructElemForegroundDataPosy, int *inGpuStructElemForegroundDataPosz, int kernelDimSize) {
//
//   int threadId = GetGlobal3dThreadIdx();
//   if(threadId < inImgDimSize) {
//     int x = *(inGpuOrigImgForegoundDataPosx + threadId);
//     int y = *(inGpuOrigImgForegoundDataPosy + threadId);
//     int z = *(inGpuOrigImgForegoundDataPosz + threadId);
//     int isEroded = Erode(x, y, z, inPaddedImg, inPaddingx, inPaddingy, inPaddingz,
//                          inGpuStructElem, inStructElemRadiusx, inStructElemRadiusy, inStructElemRadiusz,
//                          inGpuStructElemForegroundDataPosx, inGpuStructElemForegroundDataPosy, inGpuStructElemForegroundDataPosz, kernelDimSize);
//     if (isEroded) {
//       char* devPtr = (char*)inGpuOrigImg.ptr;
//       size_t pitch = inGpuOrigImg.pitch;
//       size_t slicePitch = pitch * inGpuOrigImg.ysize;
//       char* slice = devPtr + z * slicePitch;
//       unsigned char* row = (unsigned char*)(slice + y * pitch);
//       row[x] = 0;
//    }
//   }
// }
//



__host__ void CopyFromDevice3dToHostMemory(hipPitchedPtr inData, int x, int y, int z, unsigned char *&outData) {
  // hipExtent extent = make_hipExtent(x * sizeof(unsigned char), y, z);
  // hipMemcpy3DParms cpyParam = {0};
  // cpyParam.srcPtr = inData;
  // cpyParam.dstPtr = make_hipPitchedPtr( (void*)outData, x * sizeof(unsigned char), y, z);
	// cpyParam.extent = extent;
	// cpyParam.kind 	= hipMemcpyDeviceToHost;
	// hipMemcpy3D(&cpyParam);
}


__host__ void CopyFromHostToDevice3dMemory(unsigned char *inData, int x, int y, int z, hipPitchedPtr &outData) {
	// hipExtent extent = make_hipExtent(x * sizeof(unsigned char), y, z);
  // hipMalloc3D(&outData, extent);
  // hipMemcpy3DParms copyParam = {0};
  // copyParam.srcPtr = make_hipPitchedPtr( (void*)inData, x * sizeof(unsigned char), y, z );
  // copyParam.srcPtr = make_hipPitchedPtr( (void*)inData, x * sizeof(unsigned char), y, z);
	// copyParam.dstPtr = outData;
	// copyParam.extent = extent;
	// copyParam.kind 	= hipMemcpyHostToDevice;
	// hipMemcpy3D(&copyParam);
}

// __device__ unsigned char getPixel(hipPitchedPtr inImg, int x, int y, int z) {
//   // char* devPtr = (char*)inImg.ptr;
//   // size_t pitch = inImg.pitch;
//   // size_t slicePitch = pitch * inImg.ysize;
//   // char* slice = devPtr + z * slicePitch;
//   // unsigned char* row = (unsigned char*)(slice + y * pitch);
//   // return row[x];
// }

__global__ void FindBorders(hipPitchedPtr inImgData, int dimX, int dimY, int dimZ,
                            hipPitchedPtr inKernelData,
                            int radiusX, int radiusY, int radiusZ) {

  // int threadX = blockIdx.x * blockDim.x + threadIdx.x;
  // int threadY = blockIdx.y * blockDim.y + threadIdx.y;
  // int threadZ = blockIdx.z * blockDim.z + threadIdx.z;
  //
  // if (0 < threadX && dimX > threadX && 0 < threadY && dimY > threadY && 0 < threadZ && dimZ > threadZ) {
  //   unsigned char imgPixel = getPixel(inImgData, threadX, threadY, threadZ);
  //   if (imgPixel == 0) {
  //     int startX = threadX - radiusX;
  //     int startY = threadY - radiusY;
  //     int startZ = threadZ - radiusZ;
  //
  //     for (int i = startX; i < threadX + 2 * radiusX; ++i) {
  //       for (int j = startY; j < threadY + 2 * radiusY; ++j) {
  //         for (int k = startZ; k < threadZ + 2 * radiusZ; ++k) {
  //             unsigned char compPixel = getPixel(inImgData, k, j, i);
  //             // finds border pixels by comparing if adjacent pixel is foreground
  //             if(compPixel == 1) {
  //               char* devPtr = (char*)inImgData.ptr;
  //               size_t pitch = inImgData.pitch;
  //               size_t slicePitch = pitch * inImgData.ysize;
  //               char* slice = devPtr + threadZ * slicePitch;
  //               unsigned char* row = (unsigned char*)(slice + threadY * pitch);
  //               row[threadX] = 2;
  //               break;
  //             }
  //         }
  //       }
  //     }
  //   }
  // }
}



void PaintObjectAndVolImgBorderKernel(unsigned char *inImgData, int inImgDataDims[3], unsigned char *inKernelData, int inKernelRadius[3], unsigned char *&outImgData) {
  // hipPitchedPtr gpuImgData;
  // CopyFromHostToDevice3dMemory(inImgData, inImgDataDims[0], inImgDataDims[1], inImgDataDims[2], gpuImgData);
  //
  // hipPitchedPtr gpuKernelData;
  // CopyFromHostToDevice3dMemory(inKernelData, inKernelRadius[0], inKernelRadius[1], inKernelRadius[2], gpuKernelData);
  // // FindBorders(hipPitchedPtr inImgData, int dimX, int dimY, int dimZ,
  // //                             hipPitchedPtr inKernelData,
  // //                             int radiusX, int radiusY, int radiusZ)
  // int tpb0 = (inImgDataDims[0] + 8) / 8 + 1;
  // int tpb1 = (inImgDataDims[1] + 8) / 8 + 1;
  // int tpb2 = (inImgDataDims[2] + 8) / 8 + 1;
  //
  // dim3 blocks_per_grid(tpb0, tpb1, tpb2);
  // dim3 threads_per_block(8, 8, 8);
  // FindBorders<<<blocks_per_grid, threads_per_block>>>(gpuImgData, inImgDataDims[0], inImgDataDims[1], inImgDataDims[2], gpuKernelData, inKernelRadius[0], inKernelRadius[1], inKernelRadius[2]);
  // CudaCheckError();
  // outImgData = (unsigned char*)malloc(inImgDataDims[0] * inImgDataDims[1] * inImgDataDims[2] * sizeof(unsigned char));
  // CopyFromDevice3dToHostMemory(gpuImgData, inImgDataDims[0], inImgDataDims[1], inImgDataDims[2], outImgData);
  // CudaCheckError();
  // hipFree(gpuImgData.ptr);
  // hipFree(gpuKernelData.ptr);
}


__host__ void GetKernelForegroundIndex(unsigned char* inData, int inDimensions[3], std::vector<int> &x, std::vector<int> &y, std::vector<int> &z) {
    // int totalSize = inDimensions[0] * inDimensions[1] * inDimensions[2];
    // x.reserve(totalSize);
    // y.reserve(totalSize);
    // z.reserve(totalSize);
    //
    // for (int iz = 0; iz  < inDimensions[2]; ++iz) {
    //   for (int iy = 0; iy < inDimensions[1]; ++iy) {
    //     for (int ix = 0; ix < inDimensions[0]; ++ix) {
    //        unsigned char pixel = *(inData +  (iz *( inDimensions[0] * inDimensions[1] ) + (iy * inDimensions[0]) + ix));
    //        if (pixel == 1) {
    //          x.push_back(ix);
    //          y.push_back(iy);
    //          z.push_back(iz);
    //        }
    //     }
    //   }
    // }
}




__host__ void CopyFromHostToDeviceMemory(int *inData, int size, int *&outData) {
  // hipMalloc(&outData, size * sizeof(int));
  // hipMemcpy(outData, inData, size * sizeof(int), hipMemcpyHostToDevice);
}



//
//   std::vector<int> kernelPosX, kernelPosY, kernelPosZ;
//   GetKernelForegroundIndex(inKernelData, kernelDims, kernelPosX, kernelPosY, kernelPosZ);
//   int dimSize = kernelPosX.size();
//   printf("%d \n", dimSize);
//   int *gpuKernelPosX;
//   CopyFromHostToDeviceMemory(&kernelPosX[0], dimSize, gpuKernelPosX);
//   int *gpuKernelPosY;
//   CopyFromHostToDeviceMemory(&kernelPosY[0], dimSize, gpuKernelPosY);
//   int *gpuKernelPosZ;
//   CopyFromHostToDeviceMemory(&kernelPosZ[0], dimSize, gpuKernelPosZ);
//
//   int tpb0 = (inImgDataDims[0] + 8) / 8 + 1;
//   int tpb1 = (inImgDataDims[1] + 8) / 8 + 1;
//   int tpb2 = (inImgDataDims[2] + 8) / 8 + 1;
//
//   dim3 blocks_per_grid(tpb0, tpb1, tpb2);
//   dim3 threads_per_block(8, 8, 8);
//   PaintObject<<<blocks_per_grid, threads_per_block>>>(gpuImgData, inImgDataDims[0], inImgDataDims[1], inImgDataDims[2], kernelDims[0],
//       kernelDims[1], kernelDims[2], gpuKernelPosX, gpuKernelPosY, gpuKernelPosZ, dimSize, gpuOutImgData);
//   CudaCheckError();
//   outImgData = (unsigned char*)malloc(inImgDataDims[0] * inImgDataDims[1] * inImgDataDims[2] * sizeof(unsigned char));
//   CopyFromDevice3dToHostMemory(gpuOutImgData, inImgDataDims[0], inImgDataDims[1], inImgDataDims[2], outImgData);
//   CudaCheckError();
//
//   hipFree(gpuImgData.ptr);
//   hipFree(gpuOutImgData.ptr);
//   hipFree(gpuKernelPosX);
//   hipFree(gpuKernelPosY);
//   hipFree(gpuKernelPosZ);
//   hipProfilerStop();
//}


//
//
// void Erode(unsigned char *inOrigImg, int inOrigImgDims[3], unsigned char *inStructElem, int inStructElemDims[3], unsigned char *&outErodedImg) {
//
//
//
//   //
//   // std::vector<int> origImgForegoundDataPosx;
//   // std::vector<int> origImgForegoundDataPosy;
//   // std::vector<int> origImgForegoundDataPosz;
//   // GetForegroundData(inOrigImg, inOrigImgDims, origImgForegoundDataPosx, origImgForegoundDataPosy, origImgForegoundDataPosz);
//   // int imgDimSize = origImgForegoundDataPosx.size();
//   //
//   // // moves original image pos data to gpu
//   // int* gpuOrigImgForegoundDataPosx;
//   // CopyFromHostToDeviceMemory(&origImgForegoundDataPosx[0], origImgForegoundDataPosx.size(), gpuOrigImgForegoundDataPosx);
//   // int* gpuOrigImgForegoundDataPosy;
//   // CopyFromHostToDeviceMemory(&origImgForegoundDataPosy[0], origImgForegoundDataPosy.size(), gpuOrigImgForegoundDataPosy);
//   // int* gpuOrigImgForegoundDataPosz;
//   // CopyFromHostToDeviceMemory(&origImgForegoundDataPosz[0], origImgForegoundDataPosz.size(), gpuOrigImgForegoundDataPosz);
//   //
//   // // gets foreground pixel pos from kernel
//   // std::vector<int> structElemForegroundDataPosx;
//   // std::vector<int> structElemForegroundDataPosy;
//   // std::vector<int> structElemForegroundDataPosz;
//   // GetForegroundData(inStructElem, inStructElemDims, structElemForegroundDataPosx, structElemForegroundDataPosy, structElemForegroundDataPosz);
//   // int kernelDimSize = structElemForegroundDataPosx.size();
//   //
//   // // moves structuring element pos data to gpu
//     //
//   // // moves data to gpu
//   // hipPitchedPtr gpuOrigImg;
//   // CopyFromHostToDevice3dMemory(inOrigImg, inOrigImgDims[0], inOrigImgDims[1], inOrigImgDims[2], gpuOrigImg);
//   // hipPitchedPtr gpuPaddedImg;
//   // CopyFromHostToDevice3dMemory(inPaddedImg, inPaddedImgDims[0], inPaddedImgDims[1], inPaddedImgDims[2], gpuPaddedImg);
//   // hipPitchedPtr gpuStructElem;
//   // CopyFromHostToDevice3dMemory(inStructElem, inStructElemDims[0], inStructElemDims[1], inStructElemDims[2], gpuStructElem);
//   //
//   // // calculates difference of padded img and original image
//   // int padding[3];
//   // for (auto i = 0; i < 3; ++i) {
//   //     padding[i] = (inPaddedImgDims[i] - inOrigImgDims[i]) / 2;
//   // }
//   //
//   // // calculates kernel radius
//   // int kernelRadius[3];
//   // for (auto i = 0; i < 3; ++i) {
//   //     kernelRadius[i] = (inStructElemDims[i] - 1) / 2;
//   // }
//   // printf("%d", origImgForegoundDataPosz.size());
//   // int s = std::ceil((double)imgDimSize / 512);
//   // ImgErosion<<<dim3(s, 1, 1), dim3(8, 8, 8)>>>(gpuOrigImg, gpuOrigImgForegoundDataPosx, gpuOrigImgForegoundDataPosy, gpuOrigImgForegoundDataPosz, imgDimSize,
//   //                                                  gpuPaddedImg, padding[0], padding[1], padding[2],
//   //                                                  gpuStructElem, kernelRadius[0], kernelRadius[1], kernelRadius[2],
//   //                                                  gpuStructElemForegroundDataPosx, gpuStructElemForegroundDataPosy, gpuStructElemForegroundDataPosz,
//   //                                                  kernelDimSize);
//   // CudaCheckError();
//   // outErodedImg = (unsigned cha    hipDeviceSynchronize();r*)malloc(inOrigImgDims[0] * inOrigImgDims[1] * inOrigImgDims[2] * sizeof(unsigned char));
//   // CopyFromDevice3dToHostMemory(gpuOrigImg, inOrigImgDims[0], inOrigImgDims[1], inOrigImgDims[2], outErodedImg);
//   //
//   // hipFree(gpuOrigImg.ptr);
//   // hipFree(gpuPaddedImg.ptr);
//   // hipFree(gpuStructElem.ptr);
//   // hipFree(gpuStructElem.ptr);
//   // hipFree(gpuOrigImgForegoundDataPosx);
//   // hipFree(gpuOrigImgForegoundDataPosy);
//   // hipFree(gpuOrigImgForegoundDataPosz);
//   // hipFree(gpuStructElemForegroundDataPosx);
//   // hipFree(gpuStructElemForegroundDataPosy);
//   // hipFree(gpuStructElemForegroundDataPosz);
//
// }// int* gpuStructElemForegroundDataPosx;
//   // CopyFromHostToDeviceMemory(&structElemForegroundDataPosx[0], structElemForegroundDataPosx.size(), gpuStructElemForegroundDataPosx);
//   // int* gpuStructElemForegroundDataPosy;
//   // CopyFromHostToDeviceMemory(&structElemForegroundDataPosy[0], structElemForegroundDataPosy.size(), gpuStructElemForegroundDataPosy);
//   // int* gpuStructElemForegroundDataPosz;
//   // CopyFromHostToDeviceMemory(&structElemForegroundDataPosz[0], structElemForegroundDataPosz.size(), gpuStructElemForegroundDataPosz);
//   //
//   // // moves data to gpu
//   // hipPitchedPtr gpuOrigImg;
//   // CopyFromHostToDevice3dMemory(inOrigImg, inOrigImgDims[0], inOrigImgDims[1], inOrigImgDims[2], gpuOrigImg);
//   // hipPitchedPtr gpuPaddedImg;
//   // CopyFromHostToDevice3dMemory(inPaddedImg, inPaddedImgDims[0], inPaddedImgDims[1], inPaddedImgDims[2], gpuPaddedImg);
//   // hipPitchedPtr gpuStructElem;
//   // CopyFromHostToDevice3dMemory(inStructElem, inStructElemDims[0], inStructElemDims[1], inStructElemDims[2], gpuStructElem);
//   //
//   // // calculates difference of padded img and original image
//   // int padding[3];
//   // for (auto i = 0; i < 3; ++i) {
//   //     padding[i] = (inPaddedImgDims[i] - inOrigImgDims[i]) / 2;
//   // }
//   //
//   // // calculates kernel radius
//   // int kernelRadius[3];
//   // for (auto i = 0; i < 3; ++i) {
//   //     kernelRadius[i] = (inStructElemDims[i] - 1) / 2;
//   // }
//   // printf("%d", origImgForegoundDataPosz.size());
//   // int s = std::ceil((double)imgDimSize / 512);
//   // ImgErosion<<<dim3(s, 1, 1), dim3(8, 8, 8)>>>(gpuOrigImg, gpuOrigImgForegoundDataPosx, gpuOrigImgForegoundDataPosy, gpuOrigImgForegoundDataPosz, imgDimSize,
//   //                                                  gpuPaddedImg, padding[0], padding[1], padding[2],
//   //                                                  gpuStructElem, kernelRadius[0], kernelRadius[1], kernelRadius[2],
//   //                                                  gpuStructElemForegroundDataPosx, gpuStructElemForegroundDataPosy, gpuStructElemForegroundDataPosz,
//   //                                                  kernelDimSize);
//   // CudaCheckError();
//   // outErodedImg = (unsigned char*)malloc(inOrigImgDims[0] * inOrigImgDims[1] * inOrigImgDims[2] * sizeof(unsigned char));
//   // CopyFromDevice3dToHostMemory(gpuOrigImg, inOrigImgDims[0], inOrigImgDims[1], inOrigImgDims[2], outErodedImg);
//   //
//   // hipFree(gpuOrigImg.ptr);
//   // hipFree(gpuPaddedImg.ptr);
//   // hipFree(gpuStructElem.ptr);
//   // hipFree(gpuStructElem.ptr);
//   // hipFree(gpuOrigImgForegoundDataPosx);
//   // hipFree(gpuOrigImgForegoundDataPosy);
//   // hipFree(gpuOrigImgForegoundDataPosz);
//   // hipFree(gpuStructElemForegroundDataPosx);
//   // hipFree(gpuStructElemForegroundDataPosy);
//   // hipFree(gpuStructElemForegroundDataPosz);
//
// }
